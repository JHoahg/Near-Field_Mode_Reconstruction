#include "hip/hip_runtime.h"
// cuda kernel to calculate the fresnel propagation kernel.
// Johannes Hagemann 2011 - 2016
#include "/z/johannes/SciPAL/include/base/CudaComplex.h"
#
__global__
void __generate_k_element(double2* out,
                          double2* in,
                          double F,
                          size_t const height,
                          size_t const width,
                          size_t const size)

{
    typedef double NumberType;
    typedef typename SciPAL::CudaComplex<NumberType> Complex;

    //Calculate the thread ID. The thread ID determines which pixel is calculated.
    size_t index = blockDim.x*blockIdx.x+threadIdx.x;

    //Prevents kernel to calculate something outside the image vector.
    if(index<size)
    {
    size_t x = index % width - (width/2);
    size_t y = index / width - (height/2);

    NumberType kx2 = x * x * 4*M_PI*M_PI / (width * width);
    NumberType ky2 = y * y * 4*M_PI*M_PI / (height * height);
    NumberType v = - 0.5 * (kx2+ky2) / (2 * M_PI * F);
    NumberType cosv;
    NumberType sinv;
    sincos(v, &sinv, &cosv);
    Complex e(cosv, sinv); //no scaling factor, matlab fft take care of that
    //we have our value, now find position in shifted array
    size_t new_x = (x + width)%width;
    size_t new_y = (y + height)%height;
    size_t new_index = new_y * width + new_x;

    out[new_index] = toNumberType2(e * Complex(in[new_index]));
    }
}
